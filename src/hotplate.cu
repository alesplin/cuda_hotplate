/*
 * CUDA hotplate implementation
 */

#include "hotplate.h"
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[]) {
    /* stuff we'll need */
    float *oldPlate_d;
    float *newPlate_d;
    float *oldPlate_h;
    float *newPlate_h;

    oldPlate_h = (float*) calloc(PLATE_AREA, sizeof(float));
    newPlate_h = (float*) calloc(PLATE_AREA, sizeof(float));

    hipMalloc((void**) &oldPlate_d, PLATE_AREA);
    hipMalloc((void**) &newPlate_d, PLATE_AREA);

    return 0;
}

